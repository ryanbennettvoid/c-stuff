
// reference: https://gist.github.com/dpiponi/1502434

// compile: nvcc ./main.cu -o cuda

// NOTES
// - sudo apt-get install nvidia-cuda-toolkit
// - must use .cu suffix to compile properly
// - must have nVidia GPU with CUDA capabilities

#define N 1000


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void fn( int *input, int *output )
{
  int i = blockIdx.x;
  if ( i < N )
  {
    output[i] = input[i] * 2;
  }
}

int main()
{

  int hostInput[N], hostOutput[N];
  int *deviceInput, *deviceOutput;

  // allocate arrays on GPU,
  // cuda takes ownership of device pointers
  hipMalloc( (void **) &deviceInput, sizeof(int)*N );
  hipMalloc( (void **) &deviceOutput, sizeof(int)*N );

  // set input values
  for ( int i = 0; i < N; i++ )
    hostInput[i] = i;

  // copy input to GPU
  hipMemcpy( deviceInput, hostInput, sizeof(int)*N, hipMemcpyHostToDevice );

  // run GPU code on N threads- one per element
  fn<<<N, 1>>>( deviceInput, deviceOutput );

  // copy output from GPU to CPU
  hipMemcpy( hostOutput, deviceOutput, sizeof(int)*N, hipMemcpyDeviceToHost );

  for ( int i = 0; i < N; i++ )
  {
    printf( "%d\n", hostOutput[i] );
  }

  hipFree( deviceInput );
  hipFree( deviceOutput );

  return 0;
}