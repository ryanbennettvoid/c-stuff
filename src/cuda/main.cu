
// reference: https://gist.github.com/dpiponi/1502434

// compile: nvcc ./main.cu -o cuda

// NOTES
// - sudo apt-get install nvidia-cuda-toolkit
// - must use .cu suffix to compile properly
// - must have nVidia GPU with CUDA capabilities

#define N 1000


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void fn( int *input, int *output )
{
  int i = blockIdx.x;
  if ( i < N )
  {
    output[i] = input[i] * 2;
  }
}

int main()
{

  int hostInput[N], hostOutput[N];
  int *deviceInput, *deviceOutput;

  hipError_t err = hipSuccess;

  // allocate arrays on GPU,
  // cuda takes ownership of device pointers
  err = hipMalloc( (void **) &deviceInput, sizeof(int)*N );
  if ( err != hipSuccess )
  {
    fprintf( stderr, "could not allocate on GPU\n" );
  }

  err = hipMalloc( (void **) &deviceOutput, sizeof(int)*N );
  if ( err != hipSuccess )
  {
    fprintf( stderr, "could not allocate on GPU\n" );
  }

  // set input values
  for ( int i = 0; i < N; i++ )
    hostInput[i] = i;

  // copy input to GPU
  err = hipMemcpy( deviceInput, hostInput, sizeof(int)*N, hipMemcpyHostToDevice );
  if ( err != hipSuccess )
  {
    fprintf( stderr, "could not copy from hostInput to deviceInput\n" );
  }

  // run GPU code on N threads- one per element
  fn<<<N, 1>>>( deviceInput, deviceOutput );

  // copy output from GPU to CPU
  err = hipMemcpy( hostOutput, deviceOutput, sizeof(int)*N, hipMemcpyDeviceToHost );
  if ( err != hipSuccess )
  {
    fprintf( stderr, "could not copy from deviceOutput to hostOutput\n" );
  }

  for ( int i = 0; i < N; i++ )
  {
    printf( "%d\n", hostOutput[i] );
  }

  err = hipFree( deviceInput );
  if ( err != hipSuccess )
  {
    fprintf( stderr, "could not free deviceInput from GPU: %s\n", hipGetErrorString(err) );
  }

  err = hipFree( deviceOutput );
  if ( err != hipSuccess )
  {
    fprintf( stderr, "could not free deviceOutput from GPU: %s\n", hipGetErrorString(err) );
  }

  return 0;
}